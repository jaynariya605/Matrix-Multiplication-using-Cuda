#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <conio.h>
#include <iostream>
#include <hipblas.h>


// Reference : https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/
// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A,int nr_mat)
{
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A*nr_mat);
}

// using cublas matrix multiplication
 void gpu_blas_mmul(hipblasHandle_t &handle,const float *A, const float *B, float *C ,const int m, const int k, const int n,int nr_mat) {
    
	int P = nr_mat ;
	
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;
	 //create cuda stream
	hipStream_t stream[100];
	for (int i = 0; i < P; i ++)
	{
    hipStreamCreate(&stream[i]);
	}
	
	for (int iN = 0; iN < P; ++iN)
	{
	const float *const d_tmpIn = A + iN*m*k;
	//const float *const d_tmpInB = B + iN*k*n;
	float *const d_tmpOut = C + iN *m*n;
	hipblasSetStream(handle,stream[iN]); //cublas set stream
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_tmpIn, lda, B, ldb, beta, d_tmpOut, ldc);

	}
	
	
 }



int main(void)
{
clock_t tic = clock();

//C(m,k) = A(m,n)*B(n,k)

	float   *h_A ,*d_A,*h_B,*d_B,*h_C,*d_C; // intialize variable
	int M = 500 ;
	int N = 500 ; 
	int K = 400;
	int P = 100;
	int Q = 1;
	
	
	int ASize = M*N*P;
	int BSize = N*K*Q;
	int CSize = M*K*P;
	
	// Memory allocation
	h_A = (float *)malloc(sizeof(float)*ASize) ;
	h_B = (float *)malloc(sizeof(float)*BSize) ;
	h_C = (float *)malloc(sizeof(float)*CSize) ;
	
	hipMalloc((void**)&d_A, sizeof(float)*ASize) ;
	hipMalloc((void**)&d_B, sizeof(float)*BSize) ;
	hipMalloc((void**)&d_C, sizeof(float)*CSize) ;

	
	// sert intiall value 0 of matrix
	memset(h_A, 0, sizeof(float)*ASize) ;
	memset(h_B, 0, sizeof(float)*BSize) ;
	memset(h_C, 0, sizeof(float)*CSize) ;
	
    hipMemset(d_A, 0, sizeof(float)*ASize) ;
	hipMemset(d_B, 0, sizeof(float)*BSize) ;
	hipMemset(d_C, 0, sizeof(float)*CSize) ;
	
	//generate random value
	GPU_fill_rand(d_A, M, N,P ) ;
	hipMemcpy(h_A, d_A, sizeof(float)*ASize, hipMemcpyDeviceToHost) ;
	
	GPU_fill_rand(d_B, N, K,Q ) ;
	hipMemcpy(h_B, d_B, sizeof(float)*BSize, hipMemcpyDeviceToHost) ;
	
	
	
	
	

	// Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
	
	
	
	
	
	
	
	//call mulitplication function
	gpu_blas_mmul(handle,d_A, d_B, d_C, M, N, K,P);
	hipMemcpy(h_C,d_C,sizeof(float)*CSize,hipMemcpyDeviceToHost);
	
	
	
	// Destroy the handle
    hipblasDestroy(handle);
	
	
	/*//print matrix
	
	printf("==========A MAtrix================\n");
	for(int p=0;p<P;++p)
	{
	for(int i=0;i<M;++i)
	{
	for(int j=0;j<N;++j)
	{
	printf("%f ",h_A[p*N*M+j*M+i]);
	
	}
	printf("\n");
	}
	printf("\n");
	}
	
	
	
	printf("========B MAtrix========\n");
	for(int p=0;p<Q;++p)
	{
	for(int i=0;i<N;++i)
	{
	for(int j=0;j<K;++j)
	{
	printf("%f ",h_B[p*N*K+j*N+i]);
	}
	printf("\n");
	}
	printf("\n");
	}
	
	printf("========C MAtrix==========\n");
	for(int p=0;p<P;++p)
	{
	for(int i=0;i<M;i++)
	{
	for(int j=0;j<K;j++)
	{
	printf("%f ",h_C[p*M*K+i+j*M]);
	}
	printf("\n");
	}
	printf("\n");
	}*/

	clock_t toc = clock();
	printf("Cuda Runnig Time: %f seconds\n", (double)(toc - tic) / CLOCKS_PER_SEC);

}

	
	
	
	
	
	
	